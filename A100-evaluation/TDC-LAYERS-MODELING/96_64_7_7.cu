#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 1
#define TC 16
#define C 96
#define N 64
#define H 7
#define W 7

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[7];
  __shared__ float pad_temp_shared[6048];
  __shared__ float kernel_shared[1152];
  float pad_temp_shared_local[42];
  float kernel_shared_local[6];
  #pragma unroll
  for (int xx_c_init = 0; xx_c_init < 7; ++xx_c_init) {
    compute_local[(xx_c_init)] = 0.000000e+00f;
  }
  for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 216; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[((((((int)threadIdx.z) * 1512) + (((int)threadIdx.y) * 216)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (ry_outer + (((((int)threadIdx.y) * 24) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 9)) % 7))) && ((ry_outer + (((((int)threadIdx.y) * 24) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 9)) % 7)) < 8)) && (1 <= (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 9))) && ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 9) < 8)) ? data[(((((((((int)threadIdx.z) * 1176) + (((int)threadIdx.y) * 168)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 9) * 7)) + (ry_outer * 7)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 9)) - 8))] : 0.000000e+00f);
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 42; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((((int)threadIdx.y) * 14) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 96) + ((int)threadIdx.z)) < 4) {
        if ((((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 14)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) < 384) {
          if ((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 1152) {
            if (((((int)threadIdx.y) * 42) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 288) {
              if ((((((int)blockIdx.z) * 4) + (((((int)threadIdx.y) * 14) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3)) / 96)) + ((int)threadIdx.z)) < 64) {
                kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.y) * 42)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((int)blockIdx.z) * 3456) + (((int)threadIdx.z) * 864)) + (((int)threadIdx.y) * 126)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3) * 9)) + (ry_outer * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 3)))];
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 16; ++rc_inner_outer) {
      #pragma unroll
      for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
        #pragma unroll
        for (int ax1 = 0; ax1 < 6; ++ax1) {
          #pragma unroll
          for (int ax3 = 0; ax3 < 7; ++ax3) {
            pad_temp_shared_local[(((ax1 * 7) + ax3))] = pad_temp_shared[((((((rc_inner_outer * 378) + (ax1 * 63)) + (((int)threadIdx.y) * 9)) + ax3) + rx_inner_outer))];
          }
        }
        #pragma unroll
        for (int ax11 = 0; ax11 < 6; ++ax11) {
          kernel_shared_local[(ax11)] = kernel_shared[(((((((int)threadIdx.z) * 288) + (rc_inner_outer * 18)) + (ax11 * 3)) + rx_inner_outer))];
        }
        #pragma unroll
        for (int rc_inner_inner = 0; rc_inner_inner < 6; ++rc_inner_inner) {
          #pragma unroll
          for (int xx_c = 0; xx_c < 7; ++xx_c) {
            compute_local[(xx_c)] = (compute_local[(xx_c)] + (pad_temp_shared_local[(((rc_inner_inner * 7) + xx_c))] * kernel_shared_local[(rc_inner_inner)]));
          }
        }
      }
    }
  }
  #pragma unroll
  for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 7; ++xx_inner_inner_inner) {
    compute[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + xx_inner_inner_inner))] = compute_local[(xx_inner_inner_inner)];
  }
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
        __global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,1,16);

                dim3 block(1,7,4);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


