#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 4
#define TC 16
#define C 96
#define N 64
#define H 28
#define W 28

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[1920];
  __shared__ float kernel_shared[768];
  float pad_temp_shared_local[8];
  float kernel_shared_local[48];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 6; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      pad_temp_shared[((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)))] = (((((1 <= (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 18) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 18) % 120) / 30)) + ry_outer) < 29)) && (1 <= ((((int)threadIdx.x) * 18) % 30))) && (((((int)threadIdx.x) * 18) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + (((((int)threadIdx.x) * 18) / 120) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 18) % 120) / 30) * 28)) + (ry_outer * 28)) + ((((int)threadIdx.x) * 18) % 30)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 1))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 1) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 1) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 1) % 30))) && ((((((int)threadIdx.x) * 18) + 1) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 1) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 1) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 1) % 30)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 2))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 2) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 2) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 2) % 30))) && ((((((int)threadIdx.x) * 18) + 2) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 2) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 2) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 2) % 30)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 3))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 3) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 3) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 3) % 30))) && ((((((int)threadIdx.x) * 18) + 3) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 3) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 3) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 3) % 30)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 4))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 4) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 4) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 4) % 30))) && ((((((int)threadIdx.x) * 18) + 4) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 4) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 4) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 4) % 30)) - 29))] : 0.000000e+00f);
      pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 5))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 5) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 5) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 5) % 30))) && ((((((int)threadIdx.x) * 18) + 5) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 5) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 5) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 5) % 30)) - 29))] : 0.000000e+00f);
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 6) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 6) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1914) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 954) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 6))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 6) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 6) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 6) % 30))) && ((((((int)threadIdx.x) * 18) + 6) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 6) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 6) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 6) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 7) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 7) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1913) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 953) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 7))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 7) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 7) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 7) % 30))) && ((((((int)threadIdx.x) * 18) + 7) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 7) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 7) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 7) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 8) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 8) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1912) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 952) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 8))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 8) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 8) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 8) % 30))) && ((((((int)threadIdx.x) * 18) + 8) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 8) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 8) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 8) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 9) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 9) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1911) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 951) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 9))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 9) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 9) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 9) % 30))) && ((((((int)threadIdx.x) * 18) + 9) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 9) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 9) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 9) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 10) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 10) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1910) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 950) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 10))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 10) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 10) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 10) % 30))) && ((((((int)threadIdx.x) * 18) + 10) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 10) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 10) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 10) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 11) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 11) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1909) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 949) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 11))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 11) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 11) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 11) % 30))) && ((((((int)threadIdx.x) * 18) + 11) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 11) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 11) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 11) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 12) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 12) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1908) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 948) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 12))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 12) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 12) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 12) % 30))) && ((((((int)threadIdx.x) * 18) + 12) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 12) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 12) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 12) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 13) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 13) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1907) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 947) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 13))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 13) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 13) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 13) % 30))) && ((((((int)threadIdx.x) * 18) + 13) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 13) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 13) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 13) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 14) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 14) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1906) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 946) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 14))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 14) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 14) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 14) % 30))) && ((((((int)threadIdx.x) * 18) + 14) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 14) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 14) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 14) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 15) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 15) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1905) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 945) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 15))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 15) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 15) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 15) % 30))) && ((((((int)threadIdx.x) * 18) + 15) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 15) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 15) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 15) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 16) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 16) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1904) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 944) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 16))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 16) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 16) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 16) % 30))) && ((((((int)threadIdx.x) * 18) + 16) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 16) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 16) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 16) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 18) + 17) / 120)) < 16) {
        if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((((int)threadIdx.x) * 18) + 17) / 30)) < 64) {
          if ((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) < 1903) {
            if (((((int)threadIdx.y) * 240) + (((int)threadIdx.x) * 18)) < 943) {
              if (((int)threadIdx.x) < 13) {
                pad_temp_shared[(((((((int)threadIdx.z) * 960) + (((int)threadIdx.y) * 240)) + (((int)threadIdx.x) * 18)) + 17))] = (((((1 <= (((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 17) % 120) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 4) + ((((((int)threadIdx.x) * 18) + 17) % 120) / 30)) + ry_outer) < 29)) && (1 <= (((((int)threadIdx.x) * 18) + 17) % 30))) && ((((((int)threadIdx.x) * 18) + 17) % 30) < 29)) ? data[((((((((((rc_outer * 12544) + (((int)threadIdx.z) * 6272)) + (((int)threadIdx.y) * 1568)) + ((((((int)threadIdx.x) * 18) + 17) / 120) * 784)) + (((int)blockIdx.y) * 112)) + (((((((int)threadIdx.x) * 18) + 17) % 120) / 30) * 28)) + (ry_outer * 28)) + (((((int)threadIdx.x) * 18) + 17) % 30)) - 29))] : 0.000000e+00f);
              }
            }
          }
        }
      }
      kernel_shared[((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + (((((int)threadIdx.x) * 7) / 48) * 864)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) % 48) / 3) * 9)) + (ry_outer * 3)) + ((((int)threadIdx.x) * 7) % 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)) + 1))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 864)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 7) + 1) % 48) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 7) + 1) % 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)) + 2))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 864)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 7) + 2) % 48) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 7) + 2) % 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)) + 3))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 864)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 7) + 3) % 48) / 3) * 9)) + (ry_outer * 3)) + ((((int)threadIdx.x) * 7) % 3)))];
      kernel_shared[(((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)) + 4))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 864)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 7) + 4) % 48) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 7) + 1) % 3)))];
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 256) {
          if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)) < 763) {
            if (((((int)threadIdx.y) * 96) + (((int)threadIdx.x) * 7)) < 379) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)) + 5))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 864)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 7) + 5) % 48) / 3) * 9)) + (ry_outer * 3)) + (((((int)threadIdx.x) * 7) + 2) % 3)))];
              }
            }
          }
        }
      }
      if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 16) {
        if ((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + ((((int)threadIdx.x) * 7) / 3)) < 254) {
          if ((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)) < 762) {
            if (((((int)threadIdx.y) * 96) + (((int)threadIdx.x) * 7)) < 378) {
              if (((int)threadIdx.x) < 13) {
                kernel_shared[(((((((int)threadIdx.z) * 384) + (((int)threadIdx.y) * 96)) + (((int)threadIdx.x) * 7)) + 6))] = kernel[(((((((((((int)blockIdx.z) * 13824) + (((int)threadIdx.z) * 6912)) + (((int)threadIdx.y) * 1728)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 864)) + (rc_outer * 144)) + (((((((int)threadIdx.x) * 7) + 6) % 48) / 3) * 9)) + (ry_outer * 3)) + ((((int)threadIdx.x) * 7) % 3)))];
              }
            }
          }
        }
      }
      __syncthreads();
      for (int rc_inner_outer = 0; rc_inner_outer < 8; ++rc_inner_outer) {
        pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 240) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)))];
        pad_temp_shared_local[(1)] = pad_temp_shared[(((((rc_inner_outer * 240) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 1))];
        pad_temp_shared_local[(2)] = pad_temp_shared[(((((rc_inner_outer * 240) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 2))];
        pad_temp_shared_local[(3)] = pad_temp_shared[(((((rc_inner_outer * 240) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 3))];
        pad_temp_shared_local[(4)] = pad_temp_shared[(((((rc_inner_outer * 240) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 120))];
        pad_temp_shared_local[(5)] = pad_temp_shared[(((((rc_inner_outer * 240) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 121))];
        pad_temp_shared_local[(6)] = pad_temp_shared[(((((rc_inner_outer * 240) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 122))];
        pad_temp_shared_local[(7)] = pad_temp_shared[(((((rc_inner_outer * 240) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 123))];
        kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)))];
        kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 96))];
        kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 192))];
        kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 288))];
        kernel_shared_local[(24)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 384))];
        kernel_shared_local[(30)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 480))];
        kernel_shared_local[(36)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 576))];
        kernel_shared_local[(42)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 672))];
        kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 1))];
        kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 97))];
        kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 193))];
        kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 289))];
        kernel_shared_local[(25)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 385))];
        kernel_shared_local[(31)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 481))];
        kernel_shared_local[(37)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 577))];
        kernel_shared_local[(43)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 673))];
        kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 2))];
        kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 98))];
        kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 194))];
        kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 290))];
        kernel_shared_local[(26)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 386))];
        kernel_shared_local[(32)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 482))];
        kernel_shared_local[(38)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 578))];
        kernel_shared_local[(44)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 674))];
        kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 3))];
        kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 99))];
        kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 195))];
        kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 291))];
        kernel_shared_local[(27)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 387))];
        kernel_shared_local[(33)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 483))];
        kernel_shared_local[(39)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 579))];
        kernel_shared_local[(45)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 675))];
        kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 4))];
        kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 100))];
        kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 196))];
        kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 292))];
        kernel_shared_local[(28)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 388))];
        kernel_shared_local[(34)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 484))];
        kernel_shared_local[(40)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 580))];
        kernel_shared_local[(46)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 676))];
        kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 5))];
        kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 101))];
        kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 197))];
        kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 293))];
        kernel_shared_local[(29)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 389))];
        kernel_shared_local[(35)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 485))];
        kernel_shared_local[(41)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 581))];
        kernel_shared_local[(47)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 6)) + 677))];
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(12)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(18)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(24)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(30)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(36)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(42)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(6)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(12)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(18)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(24)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(30)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(36)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(42)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(13)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(19)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(25)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(31)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(37)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(43)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(7)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(13)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(19)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(25)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(31)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(37)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(43)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(14)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(20)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(26)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(32)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(38)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(44)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(2)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(8)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(14)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(20)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(26)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(32)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(38)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(44)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(3)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(9)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(15)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(21)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(27)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(33)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(39)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(45)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(3)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(9)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(15)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(21)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(27)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(33)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(39)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(45)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(4)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(10)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(16)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(22)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(28)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(34)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(40)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(46)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(4)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(10)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(16)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(22)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(28)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(34)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(40)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(46)]));
        compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(5)]));
        compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(11)]));
        compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(17)]));
        compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(23)]));
        compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(29)]));
        compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(35)]));
        compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(41)]));
        compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(47)]));
        compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(5)]));
        compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(11)]));
        compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(17)]));
        compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(23)]));
        compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(29)]));
        compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(35)]));
        compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(41)]));
        compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(47)]));
      }
    }
  }
  compute[((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1568))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 3136))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 4704))] = compute_local[(6)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 6272))] = compute_local[(8)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 7840))] = compute_local[(10)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 9408))] = compute_local[(12)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 10976))] = compute_local[(14)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1569))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 3137))] = compute_local[(5)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 4705))] = compute_local[(7)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 6273))] = compute_local[(9)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 7841))] = compute_local[(11)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 9409))] = compute_local[(13)];
  compute[(((((((((int)blockIdx.z) * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 10977))] = compute_local[(15)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 5]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,7,4);

                dim3 block(14,4,2);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<
                                   cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


