#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 1
#define TC 16
#define C 64
#define N 32
#define H 28
#define W 28

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[7];
  __shared__ float pad_temp_shared[720];
  __shared__ float kernel_shared[2304];
  float pad_temp_shared_local[14];
  float kernel_shared_local[2];
  #pragma unroll
  for (int yy_c_init = 0; yy_c_init < 7; ++yy_c_init) {
    compute_local[(yy_c_init)] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.y) * 2) + (((((int)threadIdx.z) * 23) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 3)) < 240) {
        if ((((((int)threadIdx.z) * 23) + (((int)threadIdx.y) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 720) {
          if (((((int)threadIdx.y) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 23) {
            pad_temp_shared[((((((int)threadIdx.z) * 23) + (((int)threadIdx.y) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (((((int)threadIdx.y) * 2) + (((((int)threadIdx.z) * 23) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 3)) % 30)) && ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.z) * 23) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 3)) % 30) < 29)) && (1 <= (((int)blockIdx.x) + (((((int)threadIdx.z) * 23) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 3)))) && ((((int)blockIdx.x) + (((((int)threadIdx.z) * 23) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 3)) < 29)) ? data[(((((((rc_outer * 6272) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.z) * 23) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 3)) / 30) * 784)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.z) * 23) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 3)) % 30) * 28)) + ((int)blockIdx.x)) + (((((int)threadIdx.z) * 23) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 3)) - 29))] : 0.000000e+00f);
          }
        }
      }
    }
    #pragma unroll
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 18)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((int)threadIdx.z) * 576) + (rc_outer * 72)) + (((int)threadIdx.y) * 18)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 4; ++rc_inner_outer) {
      #pragma unroll
      for (int ry_inner_outer = 0; ry_inner_outer < 3; ++ry_inner_outer) {
        #pragma unroll
        for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
          #pragma unroll
          for (int ax1 = 0; ax1 < 2; ++ax1) {
            #pragma unroll
            for (int ax2 = 0; ax2 < 7; ++ax2) {
              pad_temp_shared_local[(((ax1 * 7) + ax2))] = pad_temp_shared[(((((((rc_inner_outer * 180) + (ax1 * 90)) + (((int)threadIdx.y) * 21)) + (ax2 * 3)) + (ry_inner_outer * 3)) + rx_inner_outer))];
            }
          }
          #pragma unroll
          for (int ax11 = 0; ax11 < 2; ++ax11) {
            kernel_shared_local[(ax11)] = kernel_shared[((((((((int)threadIdx.z) * 72) + (rc_inner_outer * 18)) + (ax11 * 9)) + (ry_inner_outer * 3)) + rx_inner_outer))];
          }
          #pragma unroll
          for (int rc_inner_inner = 0; rc_inner_inner < 2; ++rc_inner_inner) {
            #pragma unroll
            for (int yy_c = 0; yy_c < 7; ++yy_c) {
              compute_local[(yy_c)] = (compute_local[(yy_c)] + (pad_temp_shared_local[(((rc_inner_inner * 7) + yy_c))] * kernel_shared_local[(rc_inner_inner)]));
            }
          }
        }
      }
    }
  }
  #pragma unroll
  for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 7; ++yy_inner_inner_inner) {
    compute[(((((((int)threadIdx.z) * 784) + (((int)threadIdx.y) * 196)) + (yy_inner_inner_inner * 28)) + ((int)blockIdx.x)))] = compute_local[(yy_inner_inner_inner)];
  }
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(28,1,1);

        dim3 block(1,4,32);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<
                                   cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


