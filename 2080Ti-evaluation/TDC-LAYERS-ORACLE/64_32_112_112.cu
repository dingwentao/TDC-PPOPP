#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 6
#define TC 32
#define C 64
#define N 32
#define H 112
#define W 112

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[1008];
  __shared__ float kernel_shared[48];
  float pad_temp_shared_local[2];
  float kernel_shared_local[8];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    if (((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 504) + ((int)threadIdx.z)) < 2) {
      if (((((int)threadIdx.z) * 18) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28)) < 36) {
        if ((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) < 1008) {
          if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) < 504) {
            if (((int)threadIdx.x) < 16) {
              pad_temp_shared[((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)))] = ((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28)) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) % 28)))) ? data[((((((((rc_outer * 25088) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28) * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) % 28)) - 113))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 504) + ((int)threadIdx.z)) < 2) {
      if (((((int)threadIdx.z) * 18) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28)) < 36) {
        if ((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) < 1007) {
          if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) < 503) {
            if (((int)threadIdx.x) < 16) {
              pad_temp_shared[(((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) + 1))] = ((((1 <= ((((int)blockIdx.y) * 16) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28))) && (((((int)blockIdx.y) * 16) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28)) < 113)) && (1 <= ((((int)blockIdx.x) * 28) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) % 28)))) ? data[((((((((rc_outer * 25088) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28) * 112)) + (((int)blockIdx.x) * 28)) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) % 28)) - 113))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 6)) < 8) {
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 3)) < 16) {
        if ((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)) < 48) {
          if (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) < 24) {
            if (((int)threadIdx.x) < 2) {
              kernel_shared[((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)))] = kernel[((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 6) * 576)) + (rc_outer * 18)) + ((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) % 6) * 3)))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 504))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 21))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 28))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 532))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 24) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 22))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 56))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 560))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 24) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    __syncthreads();
    if (((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 504) + ((int)threadIdx.z)) < 2) {
      if (((((int)threadIdx.z) * 18) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28)) < 36) {
        if ((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) < 1008) {
          if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) < 504) {
            if (((int)threadIdx.x) < 16) {
              pad_temp_shared[((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)))] = (((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28)) < 113)) ? data[((((((((rc_outer * 25088) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28) * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) % 28)) - 112))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 504) + ((int)threadIdx.z)) < 2) {
      if (((((int)threadIdx.z) * 18) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28)) < 36) {
        if ((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) < 1007) {
          if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) < 503) {
            if (((int)threadIdx.x) < 16) {
              pad_temp_shared[(((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) + 1))] = (((1 <= ((((int)blockIdx.y) * 16) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28))) && (((((int)blockIdx.y) * 16) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28)) < 113)) ? data[((((((((rc_outer * 25088) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28) * 112)) + (((int)blockIdx.x) * 28)) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) % 28)) - 112))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 6)) < 8) {
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 3)) < 16) {
        if ((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)) < 48) {
          if (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) < 24) {
            if (((int)threadIdx.x) < 2) {
              kernel_shared[((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 6) * 576)) + (rc_outer * 18)) + ((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) % 6) * 3)) + 1))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 504))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 21))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 28))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 532))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 24) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 22))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 56))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 560))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 24) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    __syncthreads();
    if (((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 504) + ((int)threadIdx.z)) < 2) {
      if (((((int)threadIdx.z) * 18) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28)) < 36) {
        if ((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) < 1008) {
          if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) < 504) {
            if (((int)threadIdx.x) < 16) {
              pad_temp_shared[((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)))] = ((((1 <= ((((int)blockIdx.y) * 16) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28))) && (((((int)blockIdx.y) * 16) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28)) < 113)) && (((((int)blockIdx.x) * 28) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) % 28)) < 111)) ? data[((((((((rc_outer * 25088) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) / 28) * 112)) + (((int)blockIdx.x) * 28)) + (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) % 28)) - 111))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 504) + ((int)threadIdx.z)) < 2) {
      if (((((int)threadIdx.z) * 18) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28)) < 36) {
        if ((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) < 1007) {
          if (((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) < 503) {
            if (((int)threadIdx.x) < 16) {
              pad_temp_shared[(((((((int)threadIdx.z) * 504) + (((int)threadIdx.y) * 32)) + (((int)threadIdx.x) * 2)) + 1))] = ((((1 <= ((((int)blockIdx.y) * 16) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28))) && (((((int)blockIdx.y) * 16) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28)) < 113)) && (((((int)blockIdx.x) * 28) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) % 28)) < 111)) ? data[((((((((rc_outer * 25088) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) / 28) * 112)) + (((int)blockIdx.x) * 28)) + ((((((int)threadIdx.y) * 32) + (((int)threadIdx.x) * 2)) + 1) % 28)) - 111))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 6)) < 8) {
      if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 3)) < 16) {
        if ((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)) < 48) {
          if (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) < 24) {
            if (((int)threadIdx.x) < 2) {
              kernel_shared[((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) / 6) * 576)) + (rc_outer * 18)) + ((((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) % 6) * 3)) + 2))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 504))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 6))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 12))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 15))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 18))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 21))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 28))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 532))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 24) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 7))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 10))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 13))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 16))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 19))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 22))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 56))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 560))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 24) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 24) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 24) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 24) + 11))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 24) + 14))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 24) + 17))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 24) + 20))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 24) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
  }
  compute[(((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 12544))] = compute_local[(1)];
  compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 25088))] = compute_local[(2)];
  compute[((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 50176)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 28)) + ((int)threadIdx.x)) + 37632))] = compute_local[(3)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 5:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 5; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 6:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 6; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 4:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 4; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 5:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 5; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 6:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 6; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[0];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[1];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 6]*data_array[1];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 6]*data_array[2];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 7]*data_array[2];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[0];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[0];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[1];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[3];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[0];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[1];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[2];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[4];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[1];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[2];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[4];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 6]*data_array[5];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 7]*data_array[2];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 7]*data_array[5];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[3];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[3];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[4];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[3];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[4];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[4];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[7];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 6]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 7]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 7]*data_array[8];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[6] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[6];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[7] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[6];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[7];
		temp_result[8] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 5]*data_array[6];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 5]*data_array[7];
		temp_result[9] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 5]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 6]*data_array[7];
		temp_result[10] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 6]*data_array[8];
		temp_result[11] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 7]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(4,7,4);

        dim3 block(28,16,2);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


